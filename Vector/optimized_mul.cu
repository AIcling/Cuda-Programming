#include <hip/hip_runtime.h>
#include <iostream>

__global__ void vectorMultiply(const float* A, const float* B, float* C, int N) {
    extern __shared__ float shared_data[]; // 共享内存声明
    float* shared_A = shared_data;         // 指向共享内存的前半部分
    float* shared_B = shared_data + blockDim.x; // 指向共享内存的后半部分

    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // 将全局内存的数据拷贝到共享内存
    if (idx < N) {
        shared_A[threadIdx.x] = A[idx];
        shared_B[threadIdx.x] = B[idx];
    }
    __syncthreads(); // 同步，确保所有线程都拷贝完数据

    // 进行向量乘法计算
    if (idx < N) {
        C[idx] = shared_A[threadIdx.x] * shared_B[threadIdx.x];
    }
}

int main() {
    int N;
    std::cout << "Enter the length of the vectors: ";
    std::cin >> N;

    size_t size = N * sizeof(float);

    // 使用cudaMallocHost分配页锁定内存
    float* h_A;
    float* h_B;
    float* h_C;
    hipHostMalloc(&h_A, size, hipHostMallocDefault);
    hipHostMalloc(&h_B, size, hipHostMallocDefault);
    hipHostMalloc(&h_C, size, hipHostMallocDefault);

    std::cout << "Enter elements of vector A:" << std::endl;
    for (int i = 0; i < N; ++i) {
        std::cin >> h_A[i];
    }

    std::cout << "Enter elements of vector B:" << std::endl;
    for (int i = 0; i < N; ++i) {
        std::cin >> h_B[i];
    }

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // 使用异步内存传输和流
    hipStream_t stream;
    hipStreamCreate(&stream);

    hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_B, h_B, size, hipMemcpyHostToDevice, stream);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // 每个线程块的共享内存大小
    size_t sharedMemSize = 2 * threadsPerBlock * sizeof(float);

    vectorMultiply<<<blocksPerGrid, threadsPerBlock, sharedMemSize, stream>>>(d_A, d_B, d_C, N);

    hipMemcpyAsync(h_C, d_C, size, hipMemcpyDeviceToHost, stream);

    // 等待所有流操作完成
    hipStreamSynchronize(stream);

    std::cout << "Result vector C:" << std::endl;
    for (int i = 0; i < N; ++i) {
        std::cout << h_C[i] << " ";
    }
    std::cout << std::endl;

    // 释放流
    hipStreamDestroy(stream);


    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);

    return 0;
}
