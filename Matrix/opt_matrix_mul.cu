#include <iostream>
#include <hip/hip_runtime.h>

#define TILE_SIZE 16

__global__ void matrixMultiplyShared(const float* A, const float* B, float* C, int N) {
    __shared__ float shared_A[TILE_SIZE][TILE_SIZE];
    __shared__ float shared_B[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;
    float value = 0.0f;

    for (int t = 0; t < (N + TILE_SIZE - 1) / TILE_SIZE; ++t) {
        if (row < N && t * TILE_SIZE + threadIdx.x < N) {
            shared_A[threadIdx.y][threadIdx.x] = A[row * N + t * TILE_SIZE + threadIdx.x];
        } else {
            shared_A[threadIdx.y][threadIdx.x] = 0.0f;
        }
        if (col < N && t * TILE_SIZE + threadIdx.y < N) {
            shared_B[threadIdx.y][threadIdx.x] = B[(t * TILE_SIZE + threadIdx.y) * N + col];
        } else {
            shared_B[threadIdx.y][threadIdx.x] = 0.0f;
        }

        __syncthreads();  

        for (int i = 0; i < TILE_SIZE; ++i) {
            value += shared_A[threadIdx.y][i] * shared_B[i][threadIdx.x];
        }

        __syncthreads();  
    }

    if (row < N && col < N) {
        C[row * N + col] = value;
    }
}

int main() {
    int N;
    std::cout << "Enter the dimension N for NxN matrices: ";
    std::cin >> N;

    float* h_A = new float[N * N];
    float* h_B = new float[N * N];
    float* h_C = new float[N * N];

    for (int i = 0; i < N * N; ++i) {
        h_A[i] = static_cast<float>(rand()) / RAND_MAX;
        h_B[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    float* d_A;
    float* d_B;
    float* d_C;
    hipMalloc((void**)&d_A, N * N * sizeof(float));
    hipMalloc((void**)&d_B, N * N * sizeof(float));
    hipMalloc((void**)&d_C, N * N * sizeof(float));

    hipMemcpy(d_A, h_A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * N * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(TILE_SIZE, TILE_SIZE);
    dim3 gridSize((N + TILE_SIZE - 1) / TILE_SIZE, (N + TILE_SIZE - 1) / TILE_SIZE);

    matrixMultiplyShared<<<gridSize, blockSize>>>(d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "Result matrix C (first 5 elements): ";
    for (int i = 0; i < 5 && i < N * N; ++i) {
        std::cout << h_C[i] << " ";
    }
    std::cout << std::endl;

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}
