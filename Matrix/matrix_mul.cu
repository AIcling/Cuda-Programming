#include <iostream>
#include <hip/hip_runtime.h>

__global__ void matrixMultiply(const float* A, const float* B, float* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        float value = 0.0f;
        for (int k = 0; k < N; ++k) {
            value += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = value;
    }
}

int main() {
    int N;
    std::cout << "Enter the dimension N for NxN matrices: ";
    std::cin >> N;

    float* h_A = new float[N * N];
    float* h_B = new float[N * N];
    float* h_C = new float[N * N];

    for (int i = 0; i < N * N; ++i) {
        h_A[i] = static_cast<float>(rand()) / RAND_MAX;
        h_B[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    float* d_A;
    float* d_B;
    float* d_C;
    hipMalloc((void**)&d_A, N * N * sizeof(float));
    hipMalloc((void**)&d_B, N * N * sizeof(float));
    hipMalloc((void**)&d_C, N * N * sizeof(float));

    hipMemcpy(d_A, h_A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * N * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (N + blockSize.y - 1) / blockSize.y);

    matrixMultiply<<<gridSize, blockSize>>>(d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "Result matrix C (first 5 elements): ";
    for (int i = 0; i < 5 && i < N * N; ++i) {
        std::cout << h_C[i] << " ";
    }
    std::cout << std::endl;

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}
